#include "hip/hip_runtime.h"

#include <unistd.h>
#include <stdio.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 1024

__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
} 

__global__ void generate( hiprandState* globalState ) 
{
    int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
 printf("ind= %d, and number =\n",ind);
    globalState[ind] = localState; 
}

int main( int argc, char** argv) 
{
    dim3 tpb(N,1,1);
    hiprandState* devStates;
    hipMalloc ( &devStates, N*sizeof( hiprandState ) );
    
    // setup seeds
    setup_kernel <<< 1, tpb >>> ( devStates, time(NULL) );

    // generate random numbers
    generate <<< 1, tpb >>> ( devStates );

    return 0;
}


